#include "hip/hip_runtime.h"
/*  Copyright 2020 Garrett Wright, Gestalt Group LLC

    This file is part of cuTWED.

    cuTWED is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    cuTWED is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with cuTWED.  If not, see <https://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdlib.h>

#include "cuTWED.h"

#define REAL_t double

#define HANDLE_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void local_distance_kernel(double A[], int nA, int degree, double DA[]){
  // implicitly assumed D can hold nA + 1 elements.
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double d;

  if( tid > nA ) return;

  if(tid == 0){
    d = 0.;
  }
  else if(tid == 1) {
    d = pow( fabs( A[tid - 1]), degree);
  }
  else {
    d = pow( fabs( A[tid - 1] - A[tid - 2] ), degree);
  }
  DA[tid] = d;
}

__global__ void dp_distance_kernel(double A[], int nA, double B[], int nB, int degree, double DP[]){
  const int tidA = blockIdx.x * blockDim.x + threadIdx.x;
  const int tidB = blockIdx.y * blockDim.y + threadIdx.y;
  const size_t tidD = tidA * (nB + 1) + tidB;
  double d;

  if(tidA >nA || tidB > nB) return;

  if(tidA==0 && tidB==0){
    d = 0;
  } else if(tidA==0 || tidB==0){
    d = INFINITY;
  }
  else{
    d = pow( fabs( A[tidA - 1] - B[tidB - 1]), degree);
    if(tidA>1 && tidB>1){
      d += pow( fabs( A[tidA - 2] - B[tidB - 2]), degree);
    }
  }

  DP[tidD] = d;
}


__global__ void evalZ_kernel(int diagIdx,
                             double DP[],
                             double DA[], int nA, double TA[],
                             double DB[], int nB, double TB[],
                             double nu, double lambda){
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // bound, consider for non square later
  if(tid > diagIdx) return;

  // map from the diagonal index and thread into the DP row/col
  const int row = tid;
  const int col = diagIdx - tid;
  if(row<1 || col <1) return;
  if(row > nA || col > nB) return;

  // get computing DP indexes out of the way
  const size_t tidD = row * (nB+1) + col;
  // lag one row
  const size_t tidDrm1 = (row-1) * (nB+1) + col;
  // lag one col
  const size_t tidDcm1 = tidD - 1;
  // lag one row and one col
  const size_t tidDrm1cm1 = tidDrm1 - 1;

  double htrans;
  double dmin;
  double dist;

  // case 1
  htrans = fabs( (double)(TA[row-1] - TB[col-1]));
  if(col>1 && row>1){
    htrans += fabs((double)(TA[row-2] - TB[col-2]));
  }
  dmin = DP[tidDrm1cm1] + DP[tidD] + nu * htrans;

  // case 2
  if(row>1)
    htrans = ((double)(TA[row-1] - TA[row-2]));
  else htrans = (double)TA[row-1];
  dist = DA[row] + DP[tidDrm1] + lambda + nu * htrans;
  // check if we need to assign new min
  if(dist<dmin){
    dmin = dist;
  }

  // case 3
  if(col>1)
    htrans = ((double)(TB[col-1] - TB[col-2]));
  else htrans = (double)TB[col-1];
  dist = DB[col] + DP[tidDcm1] + lambda + nu * htrans;
  if(dist<dmin){
    dmin = dist;
  }

  // assign result to dynamic program matrix
  DP[tidD] = dmin;
}


static void evalZ(double DP[],
           double DA[], int nA, double TA[],
           double DB[], int nB, double TB[],
           double nu, double lambda){
  int blocksz = 32;  // note this particular var might be sensitive to tuning and architectures...
  int diagIdx;
  int n;

  n = (nA+1) + (nB+1);

  for(diagIdx=1; diagIdx < n; diagIdx++){
    dim3 block_dim(blocksz);
    dim3 grid_dim((diagIdx + block_dim.x)/ block_dim.x);
    evalZ_kernel<<<grid_dim, block_dim>>>(diagIdx,DP, DA, nA, TA, DB, nB, TB, nu, lambda);
    HANDLE_ERROR(hipPeekAtLastError());
  }
}

#ifdef __cplusplus
extern "C" {
#endif
void twed_malloc_dev(int nA, double **A_dev, double  **TA_dev,
                     int nB, double **B_dev, double  **TB_dev,
                     double **DP_dev){
  //malloc on gpu and copy
  const size_t sza = (nA+1) * sizeof(**A_dev);
  HANDLE_ERROR(hipMalloc(A_dev, sza));
  HANDLE_ERROR(hipMalloc(TA_dev, sza));

  const size_t szb = (nB+1) * sizeof(**B_dev);
  HANDLE_ERROR(hipMalloc(B_dev, szb));
  HANDLE_ERROR(hipMalloc(TB_dev, szb));

  const size_t sz = (nA+1) * (nB+1) * sizeof(**DP_dev);
  HANDLE_ERROR(hipMalloc(DP_dev, sz));
}
#ifdef __cplusplus
}
#endif

#ifdef __cplusplus
extern "C" {
#endif
void twed_free_dev(double *A_dev, double  *TA_dev,
                   double *B_dev, double  *TB_dev,
                   double *DP_dev){  
  //cleanup
  HANDLE_ERROR(hipFree(A_dev));
  HANDLE_ERROR(hipFree(TA_dev));
  HANDLE_ERROR(hipFree(B_dev));
  HANDLE_ERROR(hipFree(TB_dev));
  HANDLE_ERROR(hipFree(DP_dev));
}
#ifdef __cplusplus
}
#endif

#ifdef __cplusplus
extern "C" {
#endif
void twed_copy_to_dev(int nA, double A[], double A_dev[], double TA[], double TA_dev[],
                      int nB, double B[], double B_dev[], double TB[], double TB_dev[]){ 
  const size_t sza = nA*sizeof(*A);
  HANDLE_ERROR(hipMemcpy(A_dev, A, sza, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(TA_dev, TA, sza, hipMemcpyHostToDevice));
  const size_t szb = nB*sizeof(*B);
  HANDLE_ERROR(hipMemcpy(B_dev, B , szb, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(TB_dev, TB, szb, hipMemcpyHostToDevice));
}
#ifdef __cplusplus
}
#endif


#ifdef __cplusplus
extern "C" {
#endif
double twed_dev(double A_dev[], int nA, double TA_dev[],
                double B_dev[], int nB, double TB_dev[],
                double nu, double lambda, int degree,
                double DP_dev[]){
  double *DA_dev, *DB_dev;
  double result;

  dim3 block_dim;
  dim3 grid_dim;

  const size_t sza = (nA+1) * sizeof(*A_dev);
  const size_t szb = (nB+1) * sizeof(*B_dev);
  HANDLE_ERROR(hipMalloc(&DA_dev, sza));
  HANDLE_ERROR(hipMalloc(&DB_dev, szb));

  // compute initial distance A
  block_dim.x = 256;
  grid_dim.x = (nA + block_dim.x - 1) / block_dim.x;
  local_distance_kernel<<<grid_dim, block_dim>>>(A_dev, nA, degree, DA_dev);
  HANDLE_ERROR(hipPeekAtLastError());

  // compute initial distance B
  block_dim.x = 256;
  grid_dim.x = (nB + block_dim.x - 1) / block_dim.x;
  local_distance_kernel<<<grid_dim, block_dim>>>(B_dev, nB, degree, DB_dev);
  HANDLE_ERROR(hipPeekAtLastError());

  // compute initial dynamic program matrix D
  block_dim.x = 32;
  block_dim.y = 32;
  // recall DP is nA+1 x nB+1
  grid_dim.x = (nA + block_dim.x) / block_dim.x;
  grid_dim.y = (nB + block_dim.y) / block_dim.y;
  dp_distance_kernel<<<grid_dim, block_dim>>>(A_dev, nA, B_dev, nB, degree, DP_dev);
  HANDLE_ERROR(hipPeekAtLastError());

  // iteratively update the DP matrix
  //   we process diagonals moving the diagonal from upper left to lower right,
  //         each element of a diag can is done in parallel.
  evalZ(DP_dev, DA_dev, nA, TA_dev, DB_dev, nB, TB_dev, nu, lambda);

  // the algo result should be the final distance stored in DP
  HANDLE_ERROR(hipMemcpy(&result, &DP_dev[(nA+1) * (nB+1) - 1], sizeof(result), hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipFree(DA_dev));
  HANDLE_ERROR(hipFree(DB_dev));
  
  return result;
}
#ifdef __cplusplus
}
#endif


#ifdef __cplusplus
extern "C" {
#endif
double twed(double A[], int nA, double TA[],
            double B[], int nB, double TB[],
            double nu, double lambda, int degree,
            double* DP){
  double *A_dev, *TA_dev;
  double *B_dev, *TB_dev;
  double *DP_dev;
  double result;

  // malloc gpu arrays
  twed_malloc_dev(nA, &A_dev, &TA_dev,
              nB, &B_dev, &TB_dev,
              &DP_dev);

  // copy inputs to device
  twed_copy_to_dev(nA, A, A_dev, TA, TA_dev,
                   nB, B, B_dev, TB, TB_dev);

  // compute TWED on device
  result = twed_dev(A_dev, nA, TA_dev,
                    B_dev, nB, TB_dev,
                    nu, lambda, degree,
                    DP_dev);

  // optionally copy back DP matrix
  if(DP != NULL){
    const size_t sz = (nA+1) * (nB+1) * sizeof(*DP_dev);
    HANDLE_ERROR(hipMemcpy(&result, DP_dev, sz, hipMemcpyDeviceToHost));
  }

  // free device memory
  twed_free_dev(A_dev, TA_dev,
            B_dev, TB_dev,
            DP_dev);

  return result;
}
#ifdef __cplusplus
}
#endif
